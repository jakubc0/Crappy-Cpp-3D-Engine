#include "hip/hip_runtime.h"
typedef char s8;
typedef unsigned char u8;
typedef short s16;
typedef unsigned short u16;
typedef int s32;
typedef unsigned int u32;
typedef long long s64;
typedef unsigned long long u64;
#pragma once
#include <cmath>
#include "bitmap_image.hpp"
using namespace std;

double pi = 3.1415926535;

struct vertexdat{
    int x;
    int y;
    int u;
    int v;
    int depth;
};

struct vertex{
    float x;
    float y;
    float z;
};

struct matrix{
    float m[4][4];
};

struct model{
    vertex* vertices;
    u32* faces;
    float* uv;
};

matrix mulm(matrix A, matrix B) {
    matrix result;
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            result.m[i][j] = 0;
            for (int k = 0; k < 4; ++k) {
                result.m[i][j] += A.m[i][k] * B.m[k][j];
            }
        }
    }
    return result;
}

inline int
clamp(int min, int v, int max) {
    if (v < min) return min;
    if (v > max) return max;
    return v;
}
unsigned int convertColor(int r, int g, int b) {
    r = r*65536;
    g = g*256;
    int col = r + g + b;
    return (unsigned int)col;
}

bitmap_image grass("image.bmp");

unsigned int* uploadToGPU(bitmap_image img){
    unsigned int *GPUPointer = 0;
    unsigned int color = 0x000000;
    rgb_t c;
    hipMalloc(&GPUPointer, img.width()*img.height()*sizeof(unsigned int));
    for(int i=0;i<img.width();i++){
        for(int j=0;j<img.height();j++){
            c = img.get_pixel(i, j);
            color = convertColor(c.red, c.green, c.blue);
            hipMemcpy(GPUPointer+i+j*img.width(), &color, sizeof(unsigned int), hipMemcpyHostToDevice);
        }
    }
    return GPUPointer;
}

unsigned int uvbmp(int u, int v, int z, bitmap_image image) {
    if(!image){
        return 0x000000;
    }
    int x = int((double(v)/255)*image.width());
    int y = int((double(u)/255)*image.height());
    rgb_t colour;
    image.get_pixel(x, y, colour);
    return convertColor(colour.red, colour.green, colour.blue);
}
