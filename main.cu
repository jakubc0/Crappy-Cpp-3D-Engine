#include "stuff.cu"
#include "window.cu"
#include "render.cu"
#include "stdio.h"
using namespace std;

// COMPILER COMMAND: cd "${workspaceFolder}" ; if ($?) {nvcc main.cu -o main -lGdi32 -luser32} ; if ($?) { .\main }

/*[==========================HISTORY==========================]
  |                                                           |
  |   Started around november 3rd 2023.                       |
  |   On ~december 4th this engine was capable of:            |
  |    - Drawing textured triangles.                          |
  |    - Drawing textured/untextured rectangles.              |
  |    - Drawing straight lines of a given color.             |
  |    - Filling the screen with a given color.               |
  |                                                           |
  |   On march 11th 2025 I learned to use NVIDIA CUDA.        |
  |   On march 12th 2025 I finally uploaded this to github.   |
  |   On match 16th 2025 I "converted" this project to CUDA.  |
  |                                                           |
  |   PLANS:                                                  |
  |    - Make this engine run on a GPU.                       |
  |                                                           |
*///==========================================================]

int main() {
    wiindow* window = new wiindow();
    HDC hdc = GetDC(ehwnd);

    bool running = true;
    unsigned int *gpuscr = 0;
    hipMalloc(&gpuscr, size_t(sizeof(unsigned int)*bwidth*bheight));
    hipMemcpy(gpuscr, buffermem, size_t(sizeof(unsigned int)*bwidth*bheight), hipMemcpyHostToDevice);
    unsigned int* grassimage = uploadToGPU(grass);

    while (running){
        if (!window->ProcessMessages()){
            running = false;
        }
        
        clearRect<<<bwidth, bheight>>>(gpuscr, 0x303030, 0, 0);
        drawImg<<<50, 50>>>(gpuscr, grassimage, 80, 100, 50, 50, 16, 16);
        triangle<<<142, 252>>>(gpuscr, {250, 60, 255, 0, 0}, {500, 100, 0, 255, 0}, {350, 200, 0, 0, 255});
        hipMemcpy(buffermem, gpuscr, size_t(sizeof(unsigned int)*bwidth*bheight), hipMemcpyDeviceToHost);
        StretchDIBits(hdc, 0, 0, bwidth, bheight, 0, 0, bwidth, bheight, buffermem, &bufbitinf, DIB_RGB_COLORS, SRCCOPY);
    }
    delete window;
    return 0;
}