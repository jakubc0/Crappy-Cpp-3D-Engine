#include "stuff.cu"
#include "window.cu"
#include "render.cu"
#include "stdio.h"
using namespace std;

// COMPILER COMMAND: cd "${workspaceFolder}" ; if ($?) {nvcc main.cu -o main -lGdi32 -luser32} ; if ($?) { .\main }

/*[==========================HISTORY==========================]
  |                                                           |
  |   Started around november 3rd 2023.                       |
  |   On ~december 4th this engine was capable of:            |
  |    - Drawing textured triangles.                          |
  |    - Drawing textured/untextured rectangles.              |
  |    - Drawing straight lines of a given color.             |
  |    - Filling the screen with a given color.               |
  |                                                           |
  |   On march 11th 2025 I learned to use NVIDIA CUDA.        |
  |   On march 12th 2025 I finally uploaded this to github.   |
  |   On match 16th 2025 I "converted" this project to CUDA.  |
  |                                                           |
  |   PLANS:                                                  |
  |    - Make this engine run on a GPU.                       |
  |                                                           |
*///==========================================================]

int main() {
    wiindow* window = new wiindow();
    HDC hdc = GetDC(ehwnd);

    bool running = true;
    unsigned int *gpuscr = 0;
    hipMalloc(&gpuscr, size_t(sizeof(unsigned int)*bwidth*bheight));
    hipMemcpy(gpuscr, buffermem, size_t(sizeof(unsigned int)*bwidth*bheight), hipMemcpyHostToDevice);

    while (running){
        if (!window->ProcessMessages()){
            running = false;
        }

        clrscr<<<bwidth, bheight>>>(gpuscr, 0x0000ff);
        hipMemcpy(buffermem, gpuscr, size_t(sizeof(unsigned int)*bwidth*bheight), hipMemcpyDeviceToHost);
        StretchDIBits(hdc, 0, 0, bwidth, bheight, 0, 0, bwidth, bheight, buffermem, &bufbitinf, DIB_RGB_COLORS, SRCCOPY);
    }
    delete window;
    return 0;
}