#include "hip/hip_runtime.h"
#include "stuff.cu"
#include "window.cu"
#include "render.cu"
#include "stdio.h"
using namespace std;

// COMPILER COMMAND: cd "${workspaceFolder}" ; if ($?) {nvcc main.cu -o main -O3 -lGdi32 -luser32 -rdc=true} ; if ($?) { .\main }

/*[==========================HISTORY==========================]
  |                                                           |
  |   Started around november 3rd 2023.                       |
  |   On ~december 4th this engine was capable of:            |
  |    - Drawing textured triangles.                          |
  |    - Drawing textured/untextured rectangles.              |
  |    - Drawing straight lines of a given color.             |
  |    - Filling the screen with a given color.               |
  |                                                           |
  |   On march 11th 2025 I learned to use NVIDIA CUDA.        |
  |   On march 12th 2025 I finally uploaded this to github.   |
  |   On march 16th 2025 I "converted" this project to CUDA.  |
  |                                                           |
  |   May 17th - "model" rendering (took very long)           |
  |                                                           |
  |   PLANS:                                                  |
  |                                                           |
  |                                                           |
*///==========================================================]

int main() {
    wiindow* window = new wiindow();
    HDC hdc = GetDC(ehwnd);

    bool running = true;
    unsigned int *gpuscr = 0;
    unsigned int *zbuffr = 0;
    hipMalloc(&gpuscr, size_t(sizeof(unsigned int)*bwidth*bheight));
    hipMalloc(&zbuffr, size_t(sizeof(unsigned int)*bwidth*bheight));
    hipMemcpy(gpuscr, buffermem, size_t(sizeof(unsigned int)*bwidth*bheight), hipMemcpyHostToDevice);
    unsigned int* grassimage = uploadToGPU(grass);

    vertex* cvx = 0;
    u32* ctr = 0;
    float* cuv = 0;
    
    vertex vvxx[8] = {
        {-1, -1, -1}, {-1, -1, 1},
        {-1,  1, -1}, {-1,  1, 1},
        { 1, -1, -1}, { 1, -1, 1},
        { 1,  1, -1}, { 1,  1, 1}
    };
    u32 ttrr[12*3] = {
        0, 1, 2,  1, 2, 3,
        2, 3, 6,  3, 6, 7,
        4, 5, 6,  5, 6, 7,
        0, 1, 4,  1, 4, 5,
        0, 2, 4,  2, 4, 6,
        1, 3, 5,  3, 5, 7
    };
    float uuvv[8*2] = {
        0, 0,  1, 0,
        0, 1,  1, 1,
        1, 0,  0, 0,
        1, 1,  0, 1
    };
    hipMalloc(&cvx, size_t(sizeof(vertex)*8));
    hipMalloc(&ctr, size_t(sizeof(u32)*12*3));
    hipMalloc(&cuv, size_t(sizeof(float)*8*2));
    hipMemcpy(cvx, vvxx, size_t(sizeof(vertex)*8), hipMemcpyHostToDevice);
    hipMemcpy(ctr, ttrr, size_t(sizeof(u32)*12*3), hipMemcpyHostToDevice);
    hipMemcpy(cuv, uuvv, size_t(sizeof(float)*8*2),hipMemcpyHostToDevice);
    model tri = {cvx, ctr, cuv};
    model *tttttt =0;
    hipMalloc(&tttttt, sizeof(model));
    hipMemcpy(tttttt, &tri, sizeof(model), hipMemcpyHostToDevice);

    double theta = 0;
    matrix rot = {float(std::cos(theta)), 0, float(std::sin(theta)), 0, 0, 1, 0, 0, -float(std::sin(theta)), 0, float(std::cos(theta)), 0, 0, 0, 0, 1};
    matrix pos = {1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 3, 0, 0, 0, 1};

    while (running){
        if (!window->ProcessMessages()){
            running = false;
        }
        rot = {float(std::cos(theta)), 0, float(std::sin(theta)), 0, 0, 1, 0, 0, -float(std::sin(theta)), 0, float(std::cos(theta)), 0, 0, 0, 0, 1};
        clearRect<<<bwidth, bheight>>>(gpuscr, 0x303030, 0, 0);
        clearRect<<<bwidth, bheight>>>(zbuffr, 0xffffffff, 0, 0);
        /*for(int j=0;j<12;j++) {for(int i=0;i<18;i++){
            string a = to_string(floor((((float*)buffermem)[i+j*18])*1000)/1000) + "      ";
            string b = "      ";
            for(int ii=0;ii<6;ii++){
                b[ii] = a[ii];
            }
            cout << b << " "; if(i%3==2) cout << " ";
        } cout << "\n";}cout << "\n";*/
        mdraw<<<12, 1>>>(gpuscr, mulm(pos, rot), tttttt, grassimage, 16, 16, bwidth, bheight, 100, zbuffr);
        hipMemcpy(buffermem, gpuscr, size_t(sizeof(unsigned int)*bwidth*bheight), hipMemcpyDeviceToHost);
        StretchDIBits(hdc, 0, 0, bwidth, bheight, 0, 0, bwidth, bheight, buffermem, &bufbitinf, DIB_RGB_COLORS, SRCCOPY);
        theta+=0.05;
        Sleep(10);
    }
    delete window;
    return 0;
}