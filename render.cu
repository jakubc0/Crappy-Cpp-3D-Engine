#include "hip/hip_runtime.h"
#pragma once
#include "stuff.cu"
#include "window.cu"
#include <cmath>
void cpuclearscreen(unsigned int color){
    unsigned int* pixel = (unsigned int*)buffermem;
    for (int y = 0; y < bheight; y++) {
        for (int x = 0; x < bwidth; x++){
            *pixel++ = color;}}}
void cpudrawLine(int x1, int y1, int x2, int y2, unsigned int c){
    if(y1 == y2) {
        if(x1 <= x2) {
            if(x1>=0 && x2 < bwidth){
                unsigned int* pixel = (unsigned int*)buffermem + x1 + y1*bwidth;
                for (int x = x1; x < x2; x++){
                    if(!(x<0 || x>=bwidth || y1<0 || y1>=bheight)){
                        *pixel++ = c;
                    }
                }
            }
        }
        unsigned int* pixel = (unsigned int*)buffermem + x2 + y2*bwidth;
        for (int x = x2; x < x1; x++){
            if(!(x<0 || x>=bwidth || y1<0 || y1>=bheight)){
                *pixel++ = c;
            }
        }
    }
    int ux = x1; int uy = y1; int dx = x2; int dy = y2;
    if(abs(y2-y1) > abs(x2-x1)) {
        if(y2 > y1) {ux = x2; uy = y2; dx = x1; dy = y1;}
        int xx;
        for(int y=dy;y<uy;y++){
            xx = int(ux+(dx-ux)*(double(uy-y)/double(uy-dy)));
            if(!(xx<0 || xx>=bwidth || y<0 || y>=bheight)){
                unsigned int* pixel = (unsigned int*)buffermem + xx + y*bwidth;
                *pixel = c;
            }
        }
    }else{
        if(x2 > x1) {ux = x2; uy = y2; dx = x1; dy = y1;}
        int yy;
        for(int x=dx;x<ux;x++){
            yy = int(uy+(dy-uy)*(double(ux-x)/double(ux-dx)));
            if(!(x<0 || x>=bwidth || yy<0 || yy>=bheight)){
                unsigned int* pixel = (unsigned int*)buffermem + x + yy*bwidth;
                *pixel = c;
            }
        }
    }
}
void cpudrawRect(int xpos, int ypos, int rWidth, int rHeight, unsigned int color){
    if(xpos >= bwidth || ypos >= bheight) return;
    rWidth = clamp(0,rWidth,bwidth-xpos);
    rHeight = clamp(0,rHeight,bheight-ypos);
    if(ypos < 0) {rHeight += ypos; ypos = 0;};
    if(xpos < 0) {rWidth += xpos; xpos = 0;};
    if(rWidth <=0 || rHeight <=0) return;
    for (int y = ypos; y < ypos+rHeight; y++) {
        unsigned int* pixel = (unsigned int*)buffermem + xpos + y*bwidth;
        for (int x = xpos; x < xpos+rWidth; x++){
            *pixel++ = color;}}
}
void cpudrawImg(int xpos, int ypos, int rWidth, int rHeight, bitmap_image img){
    if(xpos >= bwidth || ypos >= bheight) return;
    unsigned int color;
    rgb_t c;
    rWidth = clamp(0,rWidth,bwidth-xpos);
    rHeight = clamp(0,rHeight,bheight-ypos);
    if(ypos < 0) {rHeight += ypos; ypos = 0;};
    if(xpos < 0) {rWidth += xpos; xpos = 0;};
    if(rWidth <=0 || rHeight <=0) return;
    for (int y = ypos; y < ypos+rHeight; y++) {
        unsigned int* pixel = (unsigned int*)buffermem + xpos + y*bwidth;
        for (int x = xpos; x < xpos+rWidth; x++){
            c = img.get_pixel(int(img.width() * (double(x - xpos)/rWidth)), int(img.height() * (double(y - ypos)/rHeight)));
            color = convertColor(c.red,c.green,c.blue);
            *pixel++ = color;}}
}
void cputri(int x0, int y0, int x1, int y1, int x2, int y2, int r0, int g0, int b0, int r1, int g1, int b1, int r2, int g2, int b2, bitmap_image img){
    r0 = clamp(0,r0,255);
    g0 = clamp(0,g0,255);
    b0 = clamp(0,b0,255);
    r1 = clamp(0,r1,255);
    g1 = clamp(0,g1,255);
    b1 = clamp(0,b1,255);
    r2 = clamp(0,r2,255);
    g2 = clamp(0,g2,255);
    b2 = clamp(0,b2,255);
    if(y0 == y1 && y1 == y2){
        return;
    }
    int tstart = y0; int ustart = x0; int rstart = r0; int gstart = g0; int bstart = b0; int st = 0;
    if (y1<tstart) {tstart = y1; ustart = x1; rstart = r1; gstart = g1; bstart = b1; st = 1;}
    if (y2<tstart) {tstart = y2; ustart = x2; rstart = r2; gstart = g2; bstart = b2; st = 2;}
    int tend = y0; int uend = x0; int rend = r0; int gend = g0; int bend = b0; int nd = 0;
    if (y1>tend) {tend = y1; uend = x1; rend = r1; gend = g1; bend = b1; nd = 1;}
    if (y2>tend) {tend = y2; uend = x2; rend = r2; gend = g2; bend = b2; nd = 2;}
    int tmid = y0; int umid = x0; int rmid = r0; int gmid = g0; int bmid = b0;
    if (nd + st == 2) {tmid = y1; umid = x1; rmid = r1; gmid = g1; bmid = b1;}
    if (nd + st == 1) {tmid = y2; umid = x2; rmid = r2; gmid = g2; bmid = b2;}
    int u0, u1; //un
    int ra,rb,ga,gb,ba,bb,fr,fg,fb;
    for (int t=tstart;t<tmid;t++) {
        if(tmid == tstart){
            break;
        }
        if (t > 0 && t < bheight) {
            u0 = int((float(t - tstart)/float(tend - tstart)) * (uend - ustart));
            u1 = int((float(t - tstart)/float(tmid - tstart)) * (umid - ustart));
            ra = rstart+((float(t - tstart)/float(tend - tstart)) * (rend - rstart));
            ga = gstart+((float(t - tstart)/float(tend - tstart)) * (gend - gstart));
            ba = bstart+((float(t - tstart)/float(tend - tstart)) * (bend - bstart));
            rb = rstart+((float(t - tstart)/float(tmid - tstart)) * (rmid - rstart));
            gb = gstart+((float(t - tstart)/float(tmid - tstart)) * (gmid - gstart));
            bb = bstart+((float(t - tstart)/float(tmid - tstart)) * (bmid - bstart));
            if (u0+ustart <= u1+ustart) {
                if(u0+ustart > 0 && u1+ustart < bwidth){
                    unsigned int* pixel = (unsigned int*)buffermem + u0+ustart + t*bwidth;
                    for (int u=u0+ustart;u<u1+ustart;u++){
                        fr = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(rb-ra)+ra;
                        fg = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(gb-ga)+ga;
                        fb = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(bb-ba)+ba;
                        *pixel++ = uvbmp(fr, fg, fb, img);
                    }
                }else{
                    if(u0+ustart > 0) {
                        unsigned int* pixel = (unsigned int*)buffermem + u0+ustart + t*bwidth;
                        for (int u=u0+ustart;u<bwidth;u++){
                            fr = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(rb-ra)+ra;
                            fg = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(gb-ga)+ga;
                            fb = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(bb-ba)+ba;
                            *pixel++ = uvbmp(fr, fg, fb, img);
                        }
                    }else{
                        if(u1+ustart < bwidth) {
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<u1+ustart;u++){
                                fr = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(rb-ra)+ra;
                                fg = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(gb-ga)+ga;
                                fb = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(bb-ba)+ba;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }else{
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<bwidth;u++){
                                fr = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(rb-ra)+ra;
                                fg = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(gb-ga)+ga;
                                fb = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(bb-ba)+ba;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }
                    }
                }
            }
            if (u0+ustart > u1+ustart) {
                if (u1+ustart > 0 && u0+ustart < bwidth) {
                    unsigned int* pixel = (unsigned int*)buffermem + u1+ustart + t*bwidth;
                    for (int u=u1+ustart;u<u0+ustart;u++){
                        fr = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ra-rb)+rb;
                        fg = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ga-gb)+gb;
                        fb = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ba-bb)+bb;
                        *pixel++ = uvbmp(fr, fg, fb, img);
                    }
                }else{
                    if(u1+ustart > 0) {
                        unsigned int* pixel = (unsigned int*)buffermem + u1+ustart + t*bwidth;
                        for (int u=u1+ustart;u<bwidth;u++){
                            fr = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ra-rb)+rb;
                            fg = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ga-gb)+gb;
                            fb = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ba-bb)+bb;
                            *pixel++ = uvbmp(fr, fg, fb, img);
                        }
                    }else{
                        if(u0+ustart < bwidth) {
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<u0+ustart;u++){
                                fr = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ra-rb)+rb;
                                fg = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ga-gb)+gb;
                                fb = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ba-bb)+bb;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }else{
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<bwidth;u++){
                                fr = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ra-rb)+rb;
                                fg = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ga-gb)+gb;
                                fb = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ba-bb)+bb;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }
                    }
                }
            }
        }
    }
    for (int t=tmid;t<tend;t++) {
        if(tmid == tend){
            break;
        }
        if (t > 0 && t < bheight) {
            u0 = int((float(t - tstart)/float(tend - tstart)) * (uend - ustart));
            u1 = int((float(t - tmid)/float(tend - tmid)) * (uend - umid));
            ra = rstart+((float(t - tstart)/float(tend - tstart)) * (rend - rstart));
            ga = gstart+((float(t - tstart)/float(tend - tstart)) * (gend - gstart));
            ba = bstart+((float(t - tstart)/float(tend - tstart)) * (bend - bstart));
            rb = rmid+int((float(t - tmid)/float(tend - tmid)) * (rend - rmid));
            gb = gmid+int((float(t - tmid)/float(tend - tmid)) * (gend - gmid));
            bb = bmid+int((float(t - tmid)/float(tend - tmid)) * (bend - bmid));
            if (u0+ustart <= u1+umid) {
                if (u0 + ustart > 0 && u1 + umid < bwidth) {
                    unsigned int* pixel = (unsigned int*)buffermem + u0+ustart + t*bwidth;
                    for (int u=u0+ustart;u<u1+umid;u++){
                        fr = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(rb-ra)+ra;
                        fg = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(gb-ga)+ga;
                        fb = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(bb-ba)+ba;
                        *pixel++ = uvbmp(fr, fg, fb, img);
                    }
                }else{
                    if(u0+ustart > 0) {
                        unsigned int* pixel = (unsigned int*)buffermem + u0+ustart + t*bwidth;
                        for (int u=u0+ustart;u<bwidth;u++){
                            fr = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(rb-ra)+ra;
                            fg = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(gb-ga)+ga;
                            fb = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(bb-ba)+ba;
                            *pixel++ = uvbmp(fr, fg, fb, img);
                        }
                    }else{
                        if(u1+umid < bwidth) {
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<u1+umid;u++){
                                fr = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(rb-ra)+ra;
                                fg = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(gb-ga)+ga;
                                fb = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(bb-ba)+ba;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }else{
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<bwidth;u++){
                                fr = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(rb-ra)+ra;
                                fg = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(gb-ga)+ga;
                                fb = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(bb-ba)+ba;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }
                    }
                }
            }
            if (u0+ustart > u1+umid) {
                if (u1 + umid > 0 && u0 + ustart < bwidth) {
                    unsigned int* pixel = (unsigned int*)buffermem + u1+umid + t*bwidth;
                    for (int u=u1+umid;u<u0+ustart;u++){
                        fr = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ra-rb)+rb;
                        fg = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ga-gb)+gb;
                        fb = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ba-bb)+bb;
                        *pixel++ = uvbmp(fr, fg, fb, img);
                    }
                }else{
                    if(u1+umid > 0) {
                        unsigned int* pixel = (unsigned int*)buffermem + u1+umid + t*bwidth;
                        for (int u=u1+umid;u<bwidth;u++){
                            fr = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ra-rb)+rb;
                            fg = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ga-gb)+gb;
                            fb = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ba-bb)+bb;
                            *pixel++ = uvbmp(fr, fg, fb, img);
                        }
                    }else{
                        if(u0+ustart < bwidth) {
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<u0+ustart;u++){
                                fr = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ra-rb)+rb;
                                fg = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ga-gb)+gb;
                                fb = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ba-bb)+bb;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }else{
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<bwidth;u++){
                                fr = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ra-rb)+rb;
                                fg = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ga-gb)+gb;
                                fb = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ba-bb)+bb;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }
                    }
                }
            }
        }
    }
}

__global__ void clearRect(unsigned int* address, unsigned int color, int posX, int posY){
    unsigned int* pixel = address+blockIdx.x+posX+(threadIdx.x+posY)*640;
    if(threadIdx.x+posY<480&&blockIdx.x+posX<640&&int(blockIdx.x)+posX>=0&&int(threadIdx.x)+posY>=0) *pixel = color;
} //clearrect<<<width, height>>>(pointer, color, X, Y)

__global__ void drawImg(unsigned int* address, unsigned int* image, int posX, int posY, int W, int H, int imageW, int imageH){
    unsigned int* pixel = address+blockIdx.x+posX+(threadIdx.x+posY)*640;
    unsigned int color = image[int(float(imageW)*float(blockIdx.x)/float(W)) + int(float(imageH)*float(threadIdx.x)/float(H))*imageW];
    if(threadIdx.x+posY<480&&blockIdx.x+posX<640&&int(blockIdx.x)+posX>=0&&int(threadIdx.x)+posY>=0) *pixel = color;
} //drawImg<<<width, height>>>(pointer, image, X, Y, width, height)

__global__ void triangle(unsigned int* address, vertexdat p1, vertexdat p2, vertexdat p3){
    int x, y;
    float t, s;
    if(abs(p2.x-p1.x)>abs(p2.y-p1.y)){
        t = float(threadIdx.x)/float(abs(p2.x-p1.x)+2);
    }else{
        t = float(threadIdx.x)/float(abs(p2.y-p1.y)+2);
    }
    if(abs(p3.x-p1.x)>abs(p3.y-p1.y)){
        s = float(blockIdx.x)/float(abs(p3.x-p1.x)+2);
    }else{
        s = float(blockIdx.x)/float(abs(p3.y-p1.y)+2);
    }
    x = p1.x + int(t*float(p2.x-p1.x)) + int(s*float(p3.x-p1.x));
    y = p1.y + int(t*float(p2.y-p1.y)) + int(s*float(p3.y-p1.y));
    unsigned int* pixel = address + x + y*640;
    if(y<480&&x<640&&x>=0&&y>=0&&s+t<1) *pixel = 65536*int(p1.u*(1-s-t)+p2.u*t+p3.u*s) + 256*int(p1.v*(1-s-t)+p2.v*t+p3.v*s) + (p1.depth*(1-s-t)+p2.depth*t+p3.depth*s);
}
// the drawing file