#include "hip/hip_runtime.h"
#pragma once
#include "stuff.cu"
#include "window.cu"
#include <cmath>
void cpuclearscreen(unsigned int color){
    unsigned int* pixel = (unsigned int*)buffermem;
    for (int y = 0; y < bheight; y++) {
        for (int x = 0; x < bwidth; x++){
            *pixel++ = color;}}}
void cpudrawLine(int x1, int y1, int x2, int y2, unsigned int c){
    if(y1 == y2) {
        if(x1 <= x2) {
            if(x1>=0 && x2 < bwidth){
                unsigned int* pixel = (unsigned int*)buffermem + x1 + y1*bwidth;
                for (int x = x1; x < x2; x++){
                    if(!(x<0 || x>=bwidth || y1<0 || y1>=bheight)){
                        *pixel++ = c;
                    }
                }
            }
        }
        unsigned int* pixel = (unsigned int*)buffermem + x2 + y2*bwidth;
        for (int x = x2; x < x1; x++){
            if(!(x<0 || x>=bwidth || y1<0 || y1>=bheight)){
                *pixel++ = c;
            }
        }
    }
    int ux = x1; int uy = y1; int dx = x2; int dy = y2;
    if(abs(y2-y1) > abs(x2-x1)) {
        if(y2 > y1) {ux = x2; uy = y2; dx = x1; dy = y1;}
        int xx;
        for(int y=dy;y<uy;y++){
            xx = int(ux+(dx-ux)*(double(uy-y)/double(uy-dy)));
            if(!(xx<0 || xx>=bwidth || y<0 || y>=bheight)){
                unsigned int* pixel = (unsigned int*)buffermem + xx + y*bwidth;
                *pixel = c;
            }
        }
    }else{
        if(x2 > x1) {ux = x2; uy = y2; dx = x1; dy = y1;}
        int yy;
        for(int x=dx;x<ux;x++){
            yy = int(uy+(dy-uy)*(double(ux-x)/double(ux-dx)));
            if(!(x<0 || x>=bwidth || yy<0 || yy>=bheight)){
                unsigned int* pixel = (unsigned int*)buffermem + x + yy*bwidth;
                *pixel = c;
            }
        }
    }
}
void cpudrawRect(int xpos, int ypos, int rWidth, int rHeight, unsigned int color){
    if(xpos >= bwidth || ypos >= bheight) return;
    rWidth = clamp(0,rWidth,bwidth-xpos);
    rHeight = clamp(0,rHeight,bheight-ypos);
    if(ypos < 0) {rHeight += ypos; ypos = 0;};
    if(xpos < 0) {rWidth += xpos; xpos = 0;};
    if(rWidth <=0 || rHeight <=0) return;
    for (int y = ypos; y < ypos+rHeight; y++) {
        unsigned int* pixel = (unsigned int*)buffermem + xpos + y*bwidth;
        for (int x = xpos; x < xpos+rWidth; x++){
            *pixel++ = color;}}
}
void cpudrawImg(int xpos, int ypos, int rWidth, int rHeight, bitmap_image img){
    if(xpos >= bwidth || ypos >= bheight) return;
    unsigned int color;
    rgb_t c;
    rWidth = clamp(0,rWidth,bwidth-xpos);
    rHeight = clamp(0,rHeight,bheight-ypos);
    if(ypos < 0) {rHeight += ypos; ypos = 0;};
    if(xpos < 0) {rWidth += xpos; xpos = 0;};
    if(rWidth <=0 || rHeight <=0) return;
    for (int y = ypos; y < ypos+rHeight; y++) {
        unsigned int* pixel = (unsigned int*)buffermem + xpos + y*bwidth;
        for (int x = xpos; x < xpos+rWidth; x++){
            c = img.get_pixel(int(img.width() * (double(x - xpos)/rWidth)), int(img.height() * (double(y - ypos)/rHeight)));
            color = convertColor(c.red,c.green,c.blue);
            *pixel++ = color;}}
}
void cputri(int x0, int y0, int x1, int y1, int x2, int y2, int r0, int g0, int b0, int r1, int g1, int b1, int r2, int g2, int b2, bitmap_image img){
    r0 = clamp(0,r0,255);
    g0 = clamp(0,g0,255);
    b0 = clamp(0,b0,255);
    r1 = clamp(0,r1,255);
    g1 = clamp(0,g1,255);
    b1 = clamp(0,b1,255);
    r2 = clamp(0,r2,255);
    g2 = clamp(0,g2,255);
    b2 = clamp(0,b2,255);
    if(y0 == y1 && y1 == y2){
        return;
    }
    int tstart = y0; int ustart = x0; int rstart = r0; int gstart = g0; int bstart = b0; int st = 0;
    if (y1<tstart) {tstart = y1; ustart = x1; rstart = r1; gstart = g1; bstart = b1; st = 1;}
    if (y2<tstart) {tstart = y2; ustart = x2; rstart = r2; gstart = g2; bstart = b2; st = 2;}
    int tend = y0; int uend = x0; int rend = r0; int gend = g0; int bend = b0; int nd = 0;
    if (y1>tend) {tend = y1; uend = x1; rend = r1; gend = g1; bend = b1; nd = 1;}
    if (y2>tend) {tend = y2; uend = x2; rend = r2; gend = g2; bend = b2; nd = 2;}
    int tmid = y0; int umid = x0; int rmid = r0; int gmid = g0; int bmid = b0;
    if (nd + st == 2) {tmid = y1; umid = x1; rmid = r1; gmid = g1; bmid = b1;}
    if (nd + st == 1) {tmid = y2; umid = x2; rmid = r2; gmid = g2; bmid = b2;}
    int u0, u1; //un
    int ra,rb,ga,gb,ba,bb,fr,fg,fb;
    for (int t=tstart;t<tmid;t++) {
        if(tmid == tstart){
            break;
        }
        if (t > 0 && t < bheight) {
            u0 = int((float(t - tstart)/float(tend - tstart)) * (uend - ustart));
            u1 = int((float(t - tstart)/float(tmid - tstart)) * (umid - ustart));
            ra = rstart+((float(t - tstart)/float(tend - tstart)) * (rend - rstart));
            ga = gstart+((float(t - tstart)/float(tend - tstart)) * (gend - gstart));
            ba = bstart+((float(t - tstart)/float(tend - tstart)) * (bend - bstart));
            rb = rstart+((float(t - tstart)/float(tmid - tstart)) * (rmid - rstart));
            gb = gstart+((float(t - tstart)/float(tmid - tstart)) * (gmid - gstart));
            bb = bstart+((float(t - tstart)/float(tmid - tstart)) * (bmid - bstart));
            if (u0+ustart <= u1+ustart) {
                if(u0+ustart > 0 && u1+ustart < bwidth){
                    unsigned int* pixel = (unsigned int*)buffermem + u0+ustart + t*bwidth;
                    for (int u=u0+ustart;u<u1+ustart;u++){
                        fr = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(rb-ra)+ra;
                        fg = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(gb-ga)+ga;
                        fb = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(bb-ba)+ba;
                        *pixel++ = uvbmp(fr, fg, fb, img);
                    }
                }else{
                    if(u0+ustart > 0) {
                        unsigned int* pixel = (unsigned int*)buffermem + u0+ustart + t*bwidth;
                        for (int u=u0+ustart;u<bwidth;u++){
                            fr = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(rb-ra)+ra;
                            fg = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(gb-ga)+ga;
                            fb = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(bb-ba)+ba;
                            *pixel++ = uvbmp(fr, fg, fb, img);
                        }
                    }else{
                        if(u1+ustart < bwidth) {
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<u1+ustart;u++){
                                fr = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(rb-ra)+ra;
                                fg = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(gb-ga)+ga;
                                fb = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(bb-ba)+ba;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }else{
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<bwidth;u++){
                                fr = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(rb-ra)+ra;
                                fg = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(gb-ga)+ga;
                                fb = (float(u-(u0+ustart))/float((u1+ustart)-(u0+ustart)))*(bb-ba)+ba;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }
                    }
                }
            }
            if (u0+ustart > u1+ustart) {
                if (u1+ustart > 0 && u0+ustart < bwidth) {
                    unsigned int* pixel = (unsigned int*)buffermem + u1+ustart + t*bwidth;
                    for (int u=u1+ustart;u<u0+ustart;u++){
                        fr = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ra-rb)+rb;
                        fg = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ga-gb)+gb;
                        fb = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ba-bb)+bb;
                        *pixel++ = uvbmp(fr, fg, fb, img);
                    }
                }else{
                    if(u1+ustart > 0) {
                        unsigned int* pixel = (unsigned int*)buffermem + u1+ustart + t*bwidth;
                        for (int u=u1+ustart;u<bwidth;u++){
                            fr = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ra-rb)+rb;
                            fg = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ga-gb)+gb;
                            fb = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ba-bb)+bb;
                            *pixel++ = uvbmp(fr, fg, fb, img);
                        }
                    }else{
                        if(u0+ustart < bwidth) {
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<u0+ustart;u++){
                                fr = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ra-rb)+rb;
                                fg = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ga-gb)+gb;
                                fb = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ba-bb)+bb;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }else{
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<bwidth;u++){
                                fr = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ra-rb)+rb;
                                fg = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ga-gb)+gb;
                                fb = (float(u-(u1+ustart))/float((u0+ustart)-(u1+ustart)))*(ba-bb)+bb;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }
                    }
                }
            }
        }
    }
    for (int t=tmid;t<tend;t++) {
        if(tmid == tend){
            break;
        }
        if (t > 0 && t < bheight) {
            u0 = int((float(t - tstart)/float(tend - tstart)) * (uend - ustart));
            u1 = int((float(t - tmid)/float(tend - tmid)) * (uend - umid));
            ra = rstart+((float(t - tstart)/float(tend - tstart)) * (rend - rstart));
            ga = gstart+((float(t - tstart)/float(tend - tstart)) * (gend - gstart));
            ba = bstart+((float(t - tstart)/float(tend - tstart)) * (bend - bstart));
            rb = rmid+int((float(t - tmid)/float(tend - tmid)) * (rend - rmid));
            gb = gmid+int((float(t - tmid)/float(tend - tmid)) * (gend - gmid));
            bb = bmid+int((float(t - tmid)/float(tend - tmid)) * (bend - bmid));
            if (u0+ustart <= u1+umid) {
                if (u0 + ustart > 0 && u1 + umid < bwidth) {
                    unsigned int* pixel = (unsigned int*)buffermem + u0+ustart + t*bwidth;
                    for (int u=u0+ustart;u<u1+umid;u++){
                        fr = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(rb-ra)+ra;
                        fg = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(gb-ga)+ga;
                        fb = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(bb-ba)+ba;
                        *pixel++ = uvbmp(fr, fg, fb, img);
                    }
                }else{
                    if(u0+ustart > 0) {
                        unsigned int* pixel = (unsigned int*)buffermem + u0+ustart + t*bwidth;
                        for (int u=u0+ustart;u<bwidth;u++){
                            fr = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(rb-ra)+ra;
                            fg = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(gb-ga)+ga;
                            fb = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(bb-ba)+ba;
                            *pixel++ = uvbmp(fr, fg, fb, img);
                        }
                    }else{
                        if(u1+umid < bwidth) {
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<u1+umid;u++){
                                fr = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(rb-ra)+ra;
                                fg = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(gb-ga)+ga;
                                fb = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(bb-ba)+ba;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }else{
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<bwidth;u++){
                                fr = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(rb-ra)+ra;
                                fg = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(gb-ga)+ga;
                                fb = (float(u-(u0+ustart))/float((u1+umid)-(u0+ustart)))*(bb-ba)+ba;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }
                    }
                }
            }
            if (u0+ustart > u1+umid) {
                if (u1 + umid > 0 && u0 + ustart < bwidth) {
                    unsigned int* pixel = (unsigned int*)buffermem + u1+umid + t*bwidth;
                    for (int u=u1+umid;u<u0+ustart;u++){
                        fr = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ra-rb)+rb;
                        fg = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ga-gb)+gb;
                        fb = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ba-bb)+bb;
                        *pixel++ = uvbmp(fr, fg, fb, img);
                    }
                }else{
                    if(u1+umid > 0) {
                        unsigned int* pixel = (unsigned int*)buffermem + u1+umid + t*bwidth;
                        for (int u=u1+umid;u<bwidth;u++){
                            fr = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ra-rb)+rb;
                            fg = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ga-gb)+gb;
                            fb = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ba-bb)+bb;
                            *pixel++ = uvbmp(fr, fg, fb, img);
                        }
                    }else{
                        if(u0+ustart < bwidth) {
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<u0+ustart;u++){
                                fr = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ra-rb)+rb;
                                fg = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ga-gb)+gb;
                                fb = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ba-bb)+bb;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }else{
                            unsigned int* pixel = (unsigned int*)buffermem + t*bwidth;
                            for (int u=0;u<bwidth;u++){
                                fr = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ra-rb)+rb;
                                fg = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ga-gb)+gb;
                                fb = (float(u-(u1+umid))/float((u0+ustart)-(u1+umid)))*(ba-bb)+bb;
                                *pixel++ = uvbmp(fr, fg, fb, img);
                            }
                        }
                    }
                }
            }
        }
    }
}

__global__ void clearRect(unsigned int* address, unsigned int color, int posX, int posY){
    unsigned int* pixel = address+blockIdx.x+posX+(threadIdx.x+posY)*640;
    if(threadIdx.x+posY<480&&blockIdx.x+posX<640&&int(blockIdx.x)+posX>=0&&int(threadIdx.x)+posY>=0) *pixel = color;
} //clearrect<<<width, height>>>(pointer, color, X, Y)

__global__ void drawImg(unsigned int* address, unsigned int* image, int posX, int posY, int W, int H, int imageW, int imageH){
    unsigned int* pixel = address+blockIdx.x+posX+(threadIdx.x+posY)*640;
    unsigned int color = image[int(float(imageW)*float(blockIdx.x)/float(W)) + int(float(imageH)*float(threadIdx.x)/float(H))*imageW];
    if(threadIdx.x+posY<480&&blockIdx.x+posX<640&&int(blockIdx.x)+posX>=0&&int(threadIdx.x)+posY>=0) *pixel = color;
} //drawImg<<<width, height>>>(pointer, image, X, Y, width, height)

__global__ void triangle(unsigned int* address, vertexdat p1, vertexdat p2, vertexdat p3, unsigned int* image, unsigned int imgW, unsigned int imgH, unsigned int* zbuffer){
    int x, y, ix, iy;
    float t, s;
    if(abs(p2.x-p1.x)>abs(p2.y-p1.y)){
        t = float(threadIdx.x)/float(abs(p2.x-p1.x)+2);
    }else{
        t = float(threadIdx.x)/float(abs(p2.y-p1.y)+2);
    }
    if(abs(p3.x-p1.x)>abs(p3.y-p1.y)){
        s = float(blockIdx.x)/float(abs(p3.x-p1.x)+2);
    }else{
        s = float(blockIdx.x)/float(abs(p3.y-p1.y)+2);
    }
    x = p1.x + int(t*float(p2.x-p1.x)) + int(s*float(p3.x-p1.x));
    y = p1.y + int(t*float(p2.y-p1.y)) + int(s*float(p3.y-p1.y));
    unsigned int*  pixel = address + x + y*640;
    unsigned int* zpixel = zbuffer + x + y*640;
    unsigned int z = p1.depth + int(t*float(p2.depth-p1.depth)) + int(s*float(p3.depth-p1.depth));
    if(y<480&&x<640&&x>=0&&y>=0&&s+t<1) {
        if(z<*zpixel&&z>0){
            *zpixel = z;
            if(imgW==0x00) {
                *pixel = imgH;
            }else{
                ix = imgW*(p1.u*(1-t-s) + p2.u*t + p3.u*s)/256;
                iy = imgH*(p1.v*(1-t-s) + p2.v*t + p3.v*s)/256;
                *pixel = *(image+ix+iy*imgW);
            }
        }
    }
}

__global__ void mdraw(unsigned int* address, matrix m, model* mdl, unsigned int* image, unsigned int imgW, unsigned int imgH, u32 sw, u32 sh, float fovm, unsigned int* zbuffer){
    vertex passvert[3] = {0, 0, 0, 0, 0, 0, 0, 0, 0}; vertex passver[3] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
    passvert[0].x = (*mdl).vertices[(*mdl).faces[blockIdx.x*3]].x;
    passvert[0].y = (*mdl).vertices[(*mdl).faces[blockIdx.x*3]].y;
    passvert[0].z = (*mdl).vertices[(*mdl).faces[blockIdx.x*3]].z;
    passvert[1].x = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+1]].x;
    passvert[1].y = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+1]].y;
    passvert[1].z = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+1]].z;
    passvert[2].x = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+2]].x;
    passvert[2].y = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+2]].y;
    passvert[2].z = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+2]].z;

    /*((float*)address)[blockIdx.x*18+0] = (*mdl).vertices[(*mdl).faces[blockIdx.x*3]].x;
    ((float*)address)[blockIdx.x*18+1] = (*mdl).vertices[(*mdl).faces[blockIdx.x*3]].y;
    ((float*)address)[blockIdx.x*18+2] = (*mdl).vertices[(*mdl).faces[blockIdx.x*3]].z;
    
    ((float*)address)[blockIdx.x*18+3] = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+1]].x;
    ((float*)address)[blockIdx.x*18+4] = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+1]].y;
    ((float*)address)[blockIdx.x*18+5] = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+1]].z;

    ((float*)address)[blockIdx.x*18+6] = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+2]].x;
    ((float*)address)[blockIdx.x*18+7] = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+2]].y;
    ((float*)address)[blockIdx.x*18+8] = (*mdl).vertices[(*mdl).faces[blockIdx.x*3+2]].z;*/

    passver[0].x = m.m[0][0] * passvert[0].x + m.m[0][1] * passvert[0].y + m.m[0][2] * passvert[0].z + m.m[0][3];
    passver[0].y = m.m[1][0] * passvert[0].x + m.m[1][1] * passvert[0].y + m.m[1][2] * passvert[0].z + m.m[1][3];
    passver[0].z = m.m[2][0] * passvert[0].x + m.m[2][1] * passvert[0].y + m.m[2][2] * passvert[0].z + m.m[2][3];

    passver[1].x = m.m[0][0] * passvert[1].x + m.m[0][1] * passvert[1].y + m.m[0][2] * passvert[1].z + m.m[0][3];
    passver[1].y = m.m[1][0] * passvert[1].x + m.m[1][1] * passvert[1].y + m.m[1][2] * passvert[1].z + m.m[1][3];
    passver[1].z = m.m[2][0] * passvert[1].x + m.m[2][1] * passvert[1].y + m.m[2][2] * passvert[1].z + m.m[2][3];

    passver[2].x = m.m[0][0] * passvert[2].x + m.m[0][1] * passvert[2].y + m.m[0][2] * passvert[2].z + m.m[0][3];
    passver[2].y = m.m[1][0] * passvert[2].x + m.m[1][1] * passvert[2].y + m.m[1][2] * passvert[2].z + m.m[1][3];
    passver[2].z = m.m[2][0] * passvert[2].x + m.m[2][1] * passvert[2].y + m.m[2][2] * passvert[2].z + m.m[2][3];

    /*((float*)address)[blockIdx.x*18+0+9] = passver[0].x;
    ((float*)address)[blockIdx.x*18+1+9] = passver[0].y;
    ((float*)address)[blockIdx.x*18+2+9] = passver[0].z;
    
    ((float*)address)[blockIdx.x*18+3+9] = passver[1].x;
    ((float*)address)[blockIdx.x*18+4+9] = passver[1].y;
    ((float*)address)[blockIdx.x*18+5+9] = passver[1].z;

    ((float*)address)[blockIdx.x*18+6+9] = passver[2].x;
    ((float*)address)[blockIdx.x*18+7+9] = passver[2].y;
    ((float*)address)[blockIdx.x*18+8+9] = passver[2].z;*/

    if(passvert[0].z!=0){
        passver[0].x = (passver[0].x/passver[0].z)*fovm + sw/2;
        passver[0].y = (passver[0].y/passver[0].z)*fovm + sh/2;
    }
    if(passvert[1].z!=0){
        passver[1].x = (passver[1].x/passver[1].z)*fovm + sw/2;
        passver[1].y = (passver[1].y/passver[1].z)*fovm + sh/2;
    }
    if(passver[2].z!=0){
        passver[2].x = (passver[2].x/passver[2].z)*fovm + sw/2;
        passver[2].y = (passver[2].y/passver[2].z)*fovm + sh/2;
    }

    u32 thx;
    u32 blx;
    if(abs(passver[1].x-passver[0].x)>abs(passver[1].y-passver[0].y)){
        thx = abs(passver[1].x-passver[0].x) +2;
    }else{
        thx = abs(passver[1].y-passver[0].y) +2;
    }
    if(abs(passver[2].x-passver[0].x)>abs(passver[2].y-passver[0].y)){
        blx = abs(passver[2].x-passver[0].x) +2;
    }else{
        blx = abs(passver[2].y-passver[0].y) +2;
    }
    triangle<<<blx, thx>>>(address, 
        {int(passver[0].x), int(passver[0].y), int((*mdl).uv[(*mdl).faces[blockIdx.x*3  ]*2]*255), int((*mdl).uv[(*mdl).faces[blockIdx.x*3  ]*2+1]*255), int(passver[0].z*1000000)},
        {int(passver[1].x), int(passver[1].y), int((*mdl).uv[(*mdl).faces[blockIdx.x*3+1]*2]*255), int((*mdl).uv[(*mdl).faces[blockIdx.x*3+1]*2+1]*255), int(passver[1].z*1000000)},
        {int(passver[2].x), int(passver[2].y), int((*mdl).uv[(*mdl).faces[blockIdx.x*3+2]*2]*255), int((*mdl).uv[(*mdl).faces[blockIdx.x*3+2]*2+1]*255), int(passver[2].z*1000000)},
        image, imgW, imgH, zbuffer);
}